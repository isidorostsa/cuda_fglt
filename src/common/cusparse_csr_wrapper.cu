#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include <hipsparse.h>
#include <thrust/device_vector.h>

#include "cusparse_csr_wrapper.hpp"

#define CHECK_CUSPARSE(call)                                                                           \
    {                                                                                                  \
        hipsparseStatus_t status = call;                                                                \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                                         \
        {                                                                                              \
            fprintf(stderr, "cuSparse error %s in file '%s' in line %i : %s.\n",                       \
                    hipsparseGetErrorName(status), __FILE__, __LINE__, hipsparseGetErrorString(status)); \
            exit(1);                                                                                   \
        }                                                                                              \
    }

d_cusparse_csr::d_cusparse_csr() : d_cusparse_csr(0, 0, 0){};

d_cusparse_csr::d_cusparse_csr(size_t rows, size_t cols) : d_cusparse_csr(rows, cols, 0){};

d_cusparse_csr::d_cusparse_csr(size_t rows, size_t cols, size_t nnz) : rows(rows), cols(cols), nnz(nnz)
{

    offsets.resize(rows + 1);
    positions.resize(nnz);
    values.resize(nnz);

    update_descriptor();
}

d_cusparse_csr::d_cusparse_csr(size_t rows, size_t cols, size_t nnz, const thrust::device_vector<int> &offsets, const thrust::device_vector<int> &positions, const thrust::device_vector<float> &values)
    : rows(rows), cols(cols), nnz(nnz), offsets(offsets), positions(positions), values(values)
{
    update_descriptor();
}

d_cusparse_csr::d_cusparse_csr(size_t rows, size_t cols, size_t nnz, thrust::device_vector<int> &&offsets, thrust::device_vector<int> &&positions, thrust::device_vector<float> &&values)
    : rows(rows), cols(cols), nnz(nnz), offsets(std::move(offsets)), positions(std::move(positions)), values(std::move(values))
{
    update_descriptor();
}

d_cusparse_csr::d_cusparse_csr(size_t rows, size_t cols, size_t nnz, const thrust::host_vector<int> &values, const thrust::host_vector<int> &offsets, const thrust::host_vector<float> &positions)
    : rows(rows), cols(cols), nnz(nnz), offsets(offsets), positions(positions), values(values)
{
    update_descriptor();
}

d_cusparse_csr::~d_cusparse_csr()
{
    CHECK_CUSPARSE(hipsparseDestroySpMat(desc));
}

void d_cusparse_csr::take(size_t rows, size_t cols, size_t nnz, thrust::device_vector<int> &&offsets, thrust::device_vector<int> &&positions, thrust::device_vector<float> &&values)
{
    this->rows = rows;
    this->cols = cols;
    this->nnz = nnz;

    offsets = std::move(offsets);
    positions = std::move(positions);
    values = std::move(values);

    update_descriptor();
}

void d_cusparse_csr::update_vectors()
{
    offsets.resize(rows + 1);
    positions.resize(nnz);
    values.resize(nnz);

    update_descriptor();
}

void d_cusparse_csr::update_descriptor()
{
    assert(offsets.size() == rows + 1);
    assert(positions.size() == nnz);
    assert(values.size() == nnz);

    if (nnz == 0 || rows == 0 || cols == 0)
    {
        CHECK_CUSPARSE(
            hipsparseCreateCsr(
                &desc,
                rows,
                cols,
                0,
                NULL,
                NULL,
                NULL,
                HIPSPARSE_INDEX_32I,
                HIPSPARSE_INDEX_32I,
                HIPSPARSE_INDEX_BASE_ZERO,
                compute_type))
        return;
    }

    int64_t rows64 = static_cast<int64_t>(rows);
    int64_t cols64 = static_cast<int64_t>(cols);
    int64_t nnz64 = static_cast<int64_t>(nnz);

    CHECK_CUSPARSE(
        hipsparseCreateCsr(
            &desc,
            rows64,
            cols64,
            nnz64,
            offsets.data().get(),
            positions.data().get(),
            values.data().get(),
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            compute_type))
}

d_cusparse_csr::d_cusparse_csr(d_cusparse_csr &&other) : rows(other.rows), cols(other.cols), nnz(other.nnz),
                                                         offsets(std::move(other.offsets)), positions(std::move(other.positions)), values(std::move(other.values))

{
    update_descriptor();
}

d_cusparse_csr::d_cusparse_csr(const h_csr &h_csr_) : rows(h_csr_.rows), cols(h_csr_.cols), nnz(h_csr_.nnz),
                                                      offsets(h_csr_.offsets), positions(h_csr_.positions), values(h_csr_.values)
{
    update_descriptor();
}

// MATIX MULTIPLICATION
d_cusparse_csr d_cusparse_csr::multiply(const d_cusparse_csr &A, const d_cusparse_csr &B, hipsparseHandle_t handle)
{

    assert(A.cols == B.rows);
    assert(A.compute_type == B.compute_type);

    auto opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    hipsparseSpGEMMDescr_t spgemmDesc;
    hipsparseSpGEMM_createDescr(&spgemmDesc);

    d_cusparse_csr C(A.rows, B.cols);

    const auto compute_type = C.compute_type;

    float alpha = 1.0f;
    float beta = 0.0f;

    // inspect SpGEMM work space

    size_t bufferSize1;
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(
            handle,
            opA,
            opB,
            &alpha,
            A.get_descriptor(),
            B.get_descriptor(),
            &beta,
            C.get_descriptor(),
            compute_type,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize1,
            NULL))

    std::cout << "Megabytes needed for SpGEMM 1: " << bufferSize1 / 1024.0 / 1024.0 << " MB" << std::endl;

    // thrust::device_vector<uint8_t> d_buffer1(bufferSize1*2);

    void* d_buffer1;

    hipMalloc((void **)&d_buffer1, bufferSize1);

    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(
            handle,
            opA,
            opB,
            &alpha,
            A.get_descriptor(),
            B.get_descriptor(),
            &beta,
            C.get_descriptor(),
            compute_type,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize1,
            // d_buffer1.data().get()
            d_buffer1
        )
    )

    size_t bufferSize2 = 0;
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(
            handle,
            opA,
            opB,
            &alpha,
            A.get_descriptor(),
            B.get_descriptor(),
            &beta,
            C.get_descriptor(),
            compute_type,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize2,
            NULL
        )
    )

    std::cout << "Megabytes needed for SpGEMM 2: " << bufferSize2 / 1024.0 / 1024.0 << " MB" << std::endl;

    thrust::device_vector<char> d_buffer2(bufferSize2);

    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(
            handle,
            opA,
            opB,
            &alpha,
            A.get_descriptor(),
            B.get_descriptor(),
            &beta,
            C.get_descriptor(),
            compute_type,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize2,
            d_buffer2.data().get()))

    // now in buffer2 resides the result of the multiplication

    // !!!!!!!!!!!!!!!!!!!!!!!!
    /// IMPORTANT PART HERE!!!!
    // !!!!!!!!!!!!!!!!!!!!!!!!
    C.follow_descriptor();

    // we need to copy the result into C, but we need to resize C's vectors first
    // this will repoint the descriptor's pointers to the new empty vectors

    // now we can copy the result into the new C descriptor
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(
            handle,
            opA,
            opB,
            &alpha,
            A.get_descriptor(),
            B.get_descriptor(),
            &beta,
            C.get_descriptor(),
            compute_type,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc))

    // Now c is in the correct state, with the values from the descriptor copied into the vectors

    hipsparseSpGEMM_destroyDescr(spgemmDesc);

    return C;
}

// MATRIX-VECTOR MULTIPLICATION

thrust::device_vector<float> d_cusparse_csr::multiply(const d_cusparse_csr &A, const thrust::device_vector<float> &v, hipsparseHandle_t handle, float alpha, float beta)
{
    const size_t n = v.size();

    thrust::device_vector<float> v_out(v.size());

#if CUSPARSE_VERSION >= 12000
    hipsparseConstDnVecDescr_t v_desc;
    CHECK_CUSPARSE(hipsparseCreateConstDnVec(&v_desc, v.size(), static_cast<const void *>(v.data().get()), A.compute_type))
#else
    hipsparseDnVecDescr_t v_desc;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&v_desc, v.size(), const_cast<void *>(static_cast<const void *>(v.data().get())), A.compute_type))
#endif
    hipsparseDnVecDescr_t v_out_desc;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&v_out_desc, n, static_cast<void *>(v_out.data().get()), A.compute_type))

    size_t bufferSize = 0;

    CHECK_CUSPARSE(
        hipsparseSpMV_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha,
            A.get_descriptor(),
            v_desc,
            &beta,
            v_out_desc,
            A.compute_type,
#if CUSPARSE_VERSION >= 12000
            HIPSPARSE_SPMV_ALG_DEFAULT,
#else
            CUSPARSE_SPMV_ALG1,
#endif
            &bufferSize))

    thrust::device_vector<char> d_buffer(bufferSize);

    CHECK_CUSPARSE(
        hipsparseSpMV(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha,
            A.get_descriptor(),
            v_desc,
            &beta,
            v_out_desc,
            A.compute_type,
#if CUSPARSE_VERSION >= 12000
            HIPSPARSE_SPMV_ALG_DEFAULT,
#else
            CUSPARSE_SPMV_ALG1,
#endif
            static_cast<void *>(d_buffer.data().get())))

    hipsparseDestroyDnVec(v_desc);

    return v_out;
}

int d_cusparse_csr::get_rows() const { return rows; }
int d_cusparse_csr::get_cols() const { return cols; }
int d_cusparse_csr::get_nnz() const { return nnz; }

const thrust::device_vector<int> &d_cusparse_csr::get_offsets() const
{
    return offsets;
}

const thrust::device_vector<int> &d_cusparse_csr::get_positions() const
{
    return positions;
}

const thrust::device_vector<float> &d_cusparse_csr::get_values() const
{
    return values;
}

const hipsparseSpMatDescr_t &d_cusparse_csr::get_descriptor() const
{
    return desc;
}

void d_cusparse_csr::follow_descriptor()
{
    int64_t rows_desc, cols_desc, nnz_desc;

    CHECK_CUSPARSE(hipsparseSpMatGetSize(desc, &rows_desc, &cols_desc, &nnz_desc));

    this->rows = static_cast<size_t>(rows_desc);
    this->cols = static_cast<size_t>(cols_desc);
    this->nnz = static_cast<size_t>(nnz_desc);

    offsets.resize(rows + 1);
    positions.resize(nnz);
    values.resize(nnz);

    update_descriptor_pointers();
}

void d_cusparse_csr::resize_vectors(size_t offset_size, size_t position_value_size)
{
    this->rows = offset_size - 1;
    this->cols = position_value_size;
    this->nnz = position_value_size;

    offsets.resize(offset_size);
    positions.resize(position_value_size);
    values.resize(position_value_size);

    update_descriptor();
}

void d_cusparse_csr::update_descriptor_pointers()
{
    CHECK_CUSPARSE(hipsparseCsrSetPointers(desc, offsets.data().get(), positions.data().get(), values.data().get()))
}

// a kernel that first computes the number of nonzeros in each block
// and then copies the values into the correct positions
// __global__ void hadamard_kernel(const int n, const int A_nnz, const int B_nnz, const int *A_offsets, const int *A_positions, const float *A_values, const int *B_offsets, const int *B_positions, const float *B_values, int *C_offsets, int *C_positions, float *C_values)
// {
//     int block = blockIdx.x;
//     int tid = threadIdx.x;

//     int els_per_block = (n + gridDim.x - 1) / gridDim.x;

//     int block_start = block * els_per_block;
//     int block_end = min((block + 1) * els_per_block, n);

//     int block_size = block_end - block_start;

// }


/* cool but not used
d_cusparse_csr::operator h_csr() const
{
    thrust::host_vector<int> h_offsets(offsets);
    thrust::host_vector<int> h_positions(positions);
    thrust::host_vector<float> h_values(values);

    return h_csr(rows, cols, nnz, std::move(h_offsets), std::move(h_positions), std::move(h_values));
}
*/