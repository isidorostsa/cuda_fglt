#include <iostream>
#include <string>
#include <fstream>

#include <thrust/host_vector.h>

#include "fileio.hpp"
#include "host_structs.hpp"

#pragma GCC diagnostic ignored "-Wformat-security"
#pragma GCC diagnostic ignored "-Wunused-result"

h_coo loadFileToCoo(const std::string& filename)
{
    // check if file exists
    if(!std::ifstream(filename).good()) {
        std::cout << "File " << filename << " does not exist" << std::endl;
        exit(1);
    }

    std::ifstream fin(filename);

    int n, nnz;
    while (fin.peek() == '%')
        fin.ignore(2048, '\n');

    fin >> n >> n >> nnz;

    thrust::host_vector<int> Ai(nnz);
    thrust::host_vector<int> Aj(nnz);

    int throwaway;
    // lines may be of the form: i j or i j throwaway
    for (int i = 0; i < nnz; ++i)
    {
        fin >> Ai[i] >> Aj[i];
        Ai[i]--;
        Aj[i]--;
        if (fin.peek() != '\n')
            fin >> throwaway;
    }

    // automatically moves the vectors, no copying is done here
    return h_coo{n, nnz, std::move(Ai), std::move(Aj)};
}

h_coo loadSymmFileToCoo(const std::string& filename)
{
    std::ifstream fin(filename);
    // check if file exists
    if(!std::ifstream(filename).good()) {
        std::cout << "File " << filename << " does not exist" << std::endl;
        exit(1);
    }


    int n, nnz;
    while (fin.peek() == '%')
        fin.ignore(2048, '\n');

    fin >> n >> n >> nnz;

    thrust::host_vector<int> Ai(2*nnz);
    thrust::host_vector<int> Aj(2*nnz);

    int throwaway;
    // lines may be of the form: i j or i j throwaway
    for (int i = 0; i < nnz; i++)
    {
        int array_index = 2*i;
        fin >> Ai[array_index] >> Aj[array_index];
        Ai[array_index]--;
        Aj[array_index]--;

        if (Ai[array_index] == Aj[array_index]){
            throw std::runtime_error("Diagonal elements are not allowed");
        }

        Ai[array_index + 1] = Aj[array_index];
        Aj[array_index + 1] = Ai[array_index];

        if (fin.peek() != '\n')
            fin >> throwaway;
    }

    // automatically moves the vectors, no copying is done here
    return h_coo{n, 2*nnz, std::move(Ai), std::move(Aj)};
}

h_csr loadFileToCsr(const std::string& filename)
{
    // check file exists
    if (!std::ifstream(filename).good())
    {
        std::cout << "File " << filename << " does not exist" << std::endl;
        exit(1);
    }

    FILE *fin = fopen(filename.c_str(), "r");

    std::cout << "Opened " << filename << std::endl;

    while (fgetc(fin) == '%')
    {
        while (fgetc(fin) != '\n')
        {
            // do nothing
        };
    } // put last character back
    fseek(fin, -1, SEEK_CUR);

    int n, nnz;
    fscanf(fin, "%d %d %d", &n, &n, &nnz);
    thrust::host_vector<int> offsets(n + 1, 0);
    thrust::host_vector<int> positions(nnz);

    int i, j;
    // lines may be of the form: i j or i j throwaway where throwaway can be any number of characters until a newline
    for (int ind = 0; ind < nnz; ++ind)
    {
        fscanf(fin, "%d %d", &i, &j);
        --i;
        --j;

        positions[ind] = i;
        offsets[j + 1]++;

        // skip the rest of the line
        // unless we are at the end of the file
        if (ind < nnz - 1)
        {
            while (fgetc(fin) != '\n')
            {
            }
        }
    }

    for (int i = 0; i < n; ++i)
    {
        offsets[i + 1] += offsets[i];
    }

    // automatically moves the vectors, no copying is done here
    return h_csr(n, n, nnz, std::move(offsets), std::move(positions), thrust::host_vector<float>(nnz, 1.0f));
}

h_csr coo_to_csr(const h_coo &coo)
{

    h_csr sparse;
    sparse.rows = coo.n;
    sparse.cols = coo.n;

    sparse.nnz = coo.nnz;
    sparse.offsets.resize(coo.n + 1);
    sparse.positions.resize(coo.nnz);
    sparse.values.resize(coo.nnz);

    thrust::fill(sparse.offsets.begin(), sparse.offsets.end(), 0);
    thrust::fill(sparse.values.begin(), sparse.values.end(), 1.0f);

    for (int n = 0; n < coo.nnz; n++)
    {
        sparse.offsets[coo.Ai[n]]++;
    }

    for (int i = 0, cumsum = 0; i < coo.n; i++)
    {
        int temp = sparse.offsets[i];
        sparse.offsets[i] = cumsum;
        cumsum += temp;
    }
    sparse.offsets[coo.n] = coo.nnz;

    for (int n = 0; n < coo.nnz; n++)
    {
        int row = coo.Ai[n];
        int dest = sparse.offsets[row];

        sparse.positions[dest] = coo.Aj[n];

        sparse.offsets[row]++;
    }

    for (int i = 0, last = 0; i <= coo.n; i++)
    {
        int temp = sparse.offsets[i];
        sparse.offsets[i] = last;
        last = temp;
    }

    return sparse;
}

h_csr loadSymmFileToCsr(const std::string& filename) {
    h_coo coo = loadSymmFileToCoo(filename);
    return coo_to_csr(coo);
}