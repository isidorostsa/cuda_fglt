﻿#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cstddef>

#define ln std::cout << __LINE__ << std::endl;

#define CHECK_CUDA(call)                                               \
    {                                                                  \
        hipError_t status = (call);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CHECK_CUSPARSE(call)                                                                           \
    {                                                                                                  \
        hipsparseStatus_t status = call;                                                                \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                                         \
        {                                                                                              \
            fprintf(stderr, "cuSparse error %s in file '%s' in line %i : %s.\n",                       \
                    hipsparseGetErrorName(status), __FILE__, __LINE__, hipsparseGetErrorString(status)); \
            exit(1);                                                                                   \
        }                                                                                              \
    }

float *csrToRowMajor(int *columnsCSR, int *offsetsCSR, float *valuesCSR, int rows, int cols, int nnz)
{
    float *rowMajor = new float[rows * cols];
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            rowMajor[i * cols + j] = 0;
        }
    }

    for (int i = 0; i < rows; i++)
    {
        for (int j = offsetsCSR[i]; j < offsetsCSR[i + 1]; j++)
        {
            rowMajor[i * cols + columnsCSR[j]] = valuesCSR[j];
        }
    }

    return rowMajor;
}

void printCSR(int *columnsCSR, int *offsetsCSR, float *valuesCSR, int rows, int cols, int nnz)
{
    float *rowMajor = csrToRowMajor(columnsCSR, offsetsCSR, valuesCSR, rows, cols, nnz);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            std::cout << std::setw(5) << rowMajor[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

template<typename T>
std::ostream& operator<<(std::ostream& os, const thrust::host_vector<T>& vec)
{
    os << "|";
    for (const T& el : vec) {
        os << " " << el << " |";
    } os << std::endl;
    return os;
}

int main(int argc, char *argv[])
{
    // HOST DATA
    const int A_rows = 4;
    const int A_cols = 4;
    const int A_nnz = 9;

    int A_offsets[] = {0, 3, 4, 7, 9};
    int A_columns[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};
    float A_values[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};

    thrust::host_vector<int> h_A_offs(A_offsets, A_offsets + 5);
    std::cout << h_A_offs;

    thrust::host_vector<int> h_A_cols(A_columns, A_columns + 9);
    std::cout << h_A_cols;

    thrust::host_vector<float> h_A_vals(A_values, A_values + 9);
    std::cout << h_A_vals;

    std::cout << "A = " << std::endl;
    printCSR(h_A_cols.data(), h_A_offs.data(), h_A_vals.data(), A_rows, A_cols, A_nnz);

    const int64_t A2_rows = A_rows;
    const int64_t A2_cols = A_cols;
    int64_t A2_nnz = 0;

    thrust::host_vector<int> h_A2_offs;
    thrust::host_vector<int> h_A2_cols;
    thrust::host_vector<float> h_A2_vals;

    // DEVICE DATA
    thrust::device_vector<int> d_A_offs(h_A_offs);
    thrust::device_vector<int> d_A_cols(h_A_cols);
    thrust::device_vector<float> d_A_vals(h_A_vals);

    thrust::device_vector<int> d_A2_offs(h_A2_offs);
    thrust::device_vector<int> d_A2_cols;
    thrust::device_vector<float> d_A2_vals;

    // CUSPARSE STBRTS HERE
    const hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipDataType computeType = HIP_R_32F;

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    CHECK_CUSPARSE(cusparseLoggerSetMask(1 | 0 | 4 | 8 | 16))

    hipsparseSpMatDescr_t A_CSR;
    CHECK_CUSPARSE(
        hipsparseCreateCsr(
            &A_CSR,
            A_rows,
            A_cols,
            A_nnz,
            d_A_offs.data().get(),
            d_A_cols.data().get(),
            d_A_vals.data().get(),
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            computeType
        )
    )

    hipsparseSpMatDescr_t A2_CSR;
    CHECK_CUSPARSE(
        hipsparseCreateCsr(
            &A2_CSR,
            A2_rows,
            A2_cols,
            0,
            NULL,
            NULL,
            NULL,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            computeType))

    // spGEMM computation:
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc))

    // void *buffer1_d = NULL, *buffer2_d = NULL;
    //  in thrust dialect:

    float alpha = 1.0f, beta = 0.0f;

    size_t bufferSize1 = 0;
    // estimate memmory needed for this
    // PASSING NULL AS THE LAST PARAMETER
    // TELLS CUSPARSE TO HANDLE MEMMORY IN ITS OWN BUFFER
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(
            handle,
            opA,
            opB,
            &alpha,
            A_CSR,
            A_CSR,
            &beta,
            A2_CSR,
            computeType,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize1,
            NULL
        )
    )
    
    thrust::device_vector<uint8_t> d_buffer1(bufferSize1);

    std::cout
        << "bufferSize1_d: " << bufferSize1 << std::endl;

    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(
            handle,
            opA,
            opB,
            &alpha,
            A_CSR,
            A_CSR,
            &beta,
            A2_CSR,
            computeType,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize1,
            d_buffer1.data().get()
        )
    )

    size_t bufferSize2 = 0;
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(
            handle,
            opA,
            opB,
            &alpha,
            A_CSR,
            A_CSR,
            &beta,
            A2_CSR,
            computeType,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize2,
            NULL
        )
    )

    std::cout << "bufferSize2_d: " << bufferSize2 << std::endl;
    thrust::device_vector<uint8_t> d_buffer2(bufferSize2);

    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(
            handle,
            opA,
            opB,
            &alpha,
            A_CSR,
            A_CSR,
            &beta,
            A2_CSR,
            computeType,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc,
            &bufferSize2,
            d_buffer2.data().get()
        )
    )

    int64_t throwaway;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(A2_CSR, &throwaway, &throwaway, &A2_nnz));

    std::cout << "A2_rows: " << A2_rows << std::endl;
    std::cout << "A2_cols: " << A2_cols << std::endl;
    std::cout << "A2_nnz: " << A2_nnz << std::endl;

    d_A2_offs.resize(A2_rows + 1);
    d_A2_vals.resize(A2_nnz);
    d_A2_cols.resize(A2_nnz);

    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(
            A2_CSR,
            d_A2_offs.data().get(),
            d_A2_cols.data().get(),
            d_A2_vals.data().get()
        )
    )

    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(
            handle,
            opA,
            opB,
            &alpha,
            A_CSR,
            A_CSR,
            &beta,
            A2_CSR,
            computeType,
            HIPSPARSE_SPGEMM_DEFAULT,
            spgemmDesc
        )
    )

    // IT IS DONE
    CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(spgemmDesc));
    CHECK_CUDA(hipDeviceSynchronize());

    // Now we will calculate A2 * [1 1 .... 1]^T
    thrust::host_vector<float> h_mulVec(A2_rows, 1.0f);
    thrust::device_vector<float> d_mulVec(h_mulVec);

    hipsparseDnVecDescr_t mulVec_descr;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&mulVec_descr, A_rows, d_mulVec.data().get(), HIP_R_32F))

    thrust::device_vector<float> h_resVec(A2_rows);
    thrust::device_vector<float> d_resVec(A2_rows);
    hipsparseDnVecDescr_t resVec_descr;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&resVec_descr, A_rows, d_resVec.data().get(), HIP_R_32F))

    size_t bufferSize = 0;
    CHECK_CUSPARSE(
        hipsparseSpMV_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha,
            A2_CSR,
            mulVec_descr,
            &beta,
            resVec_descr,
            HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT,
            &bufferSize
        )
    )

    thrust::device_vector<uint8_t> d_buffer(bufferSize);

    CHECK_CUSPARSE(
        hipsparseSpMV(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha,
            A2_CSR,
            mulVec_descr,
            &beta,
            resVec_descr,
            HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT,
            d_buffer.data().get()
        )
    )

    CHECK_CUSPARSE(hipsparseDestroySpMat(A_CSR));
    CHECK_CUSPARSE(hipsparseDestroySpMat(A2_CSR));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    h_A2_offs.resize(A2_rows + 1);
    thrust::copy(d_A2_offs.begin(), d_A2_offs.end(), h_A2_offs.begin());

    h_A2_cols.resize(A2_nnz);
    thrust::copy(d_A2_cols.begin(), d_A2_cols.end(), h_A2_cols.begin());

    h_A2_vals.resize(A2_nnz);
    thrust::copy(d_A2_vals.begin(), d_A2_vals.end(), h_A2_vals.begin());

    std::cout << "A2 = " << std::endl;
    printCSR(h_A2_cols.data(), h_A2_offs.data(), h_A2_vals.data(), A_rows, A2_cols, A2_nnz);

    std::cout << "A2 * [1 1 ... 1]^T = ";

    h_resVec.resize(A2_rows);
    thrust::copy(d_resVec.begin(), d_resVec.end(), h_resVec.begin());
    for (int i = 0; i < A_rows; i++)
    {
        std::cout << h_resVec[i] << ", ";
    } std::cout << std::endl;

    return 0;
}
