#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include "common/fileio.hpp"
#include "common/printing.hpp"
#include "common/device_csr.hpp"

#define time_op(name, op) \
        start = std::chrono::high_resolution_clock::now(); \
        op; \
        end = std::chrono::high_resolution_clock::now(); \
        std::cout << name << " took " << std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count() << " ms" << std::endl;

struct d3_trans
{
    __thrust_exec_check_disable__
        __host__ __device__ constexpr COMPUTE_TYPE
        operator()(const COMPUTE_TYPE &lhs, const COMPUTE_TYPE &rhs) const
    {
        return (lhs) * (lhs - 1) / 2 -  rhs;
    }
};

struct d2_trans
{
    __thrust_exec_check_disable__
        __host__ __device__ constexpr COMPUTE_TYPE
        operator()(const COMPUTE_TYPE &lhs, const COMPUTE_TYPE &rhs) const
    {
        return (lhs) - 2 * rhs;
    }
};

__global__ void c3_kernel(const int *offsets, const int *positions, COMPUTE_TYPE *c3, int n)
{    
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
  {
    const int i_nb_start = offsets[i];
    const int i_nb_end = offsets[i + 1];

    int amt_i = 0;
    for (int i_nb_idx = i_nb_start; i_nb_idx < i_nb_end; i_nb_idx++)
    {
      const int j = positions[i_nb_idx];
      if (i < j) break;

      const int j_nb_start = offsets[j];
      const int j_nb_end = offsets[j + 1];

      int _i_nb_idx = i_nb_start;
      int _j_nb_idx = j_nb_start;

      int amt_j = 0;
      while (_i_nb_idx < i_nb_end && _j_nb_idx < j_nb_end)
      {
        const int _i_nb_pos = positions[_i_nb_idx];
        const int _j_nb_pos = positions[_j_nb_idx];

        if(_i_nb_pos > i || _j_nb_pos > j) break;

        if (_i_nb_pos > _j_nb_pos)
        {
          _j_nb_idx++;
        }
        else if (_i_nb_pos < _j_nb_pos)
        {
          _i_nb_idx++;
        }
        else
        {
          amt_j++, atomicAdd(&c3[_i_nb_pos], 1),  _i_nb_idx++, _j_nb_idx++;
        }
      }

      amt_i += amt_j;

      atomicAdd(&c3[j], amt_j);
    }

    atomicAdd(&c3[i], amt_i);
  }
}

thrust::device_vector<COMPUTE_TYPE> get_c3_v3(const d_csr& A) {
    const int n = A.get_rows();

    thrust::device_vector<COMPUTE_TYPE> d_c3(n);

    const int *d_A_offs = A.get_offsets().data().get();
    const int *d_A_pos = A.get_positions().data().get();

    COMPUTE_TYPE *d_c3_ptr = d_c3.data().get();

    const int SmSize = 16;
    const int threadsPerBlock = 256;
    const int threadsPerSM = threadsPerBlock * SmSize;

    const int FullSMs = (n + threadsPerSM - 1) / threadsPerSM;

    c3_kernel<<<SmSize*FullSMs, threadsPerBlock>>>(d_A_offs, d_A_pos, d_c3_ptr, n);
    // c3_kernel<<<1, 1>>>(d_A_offs, d_A_pos, d_c3_ptr, n);
    hipDeviceSynchronize();

    return d_c3;
}

thrust::host_vector<
    thrust::device_vector<COMPUTE_TYPE>
> fglt(const d_csr& d_A) {

    std::chrono::system_clock::time_point start;
    std::chrono::system_clock::time_point end;

    const int n = d_A.get_rows();
    /*
                CALCULATE p1
    */
time_op("p1",
    thrust::device_vector<COMPUTE_TYPE> d_p1(n);
    // adjecent difference
    thrust::transform(
        d_A.offsets.begin() + 1, d_A.offsets.end(),
        d_A.offsets.begin(), // d_A_offs_COMPUTE_TYPE.end()-1,
        d_p1.begin(),
        thrust::minus<COMPUTE_TYPE>()
    )
)
    /*
                CALCULATE c3
    */

time_op("c3",
    thrust::device_vector<COMPUTE_TYPE> d_c3 = get_c3_v3(d_A);
)

    /*
                CALCULATE p2
    */
time_op("p2",
    thrust::device_vector<COMPUTE_TYPE> d_Ap1 = d_csr::spmv_symbolic(d_A, d_p1);
    thrust::device_vector<COMPUTE_TYPE> d_p2(n);
    thrust::transform(
        d_Ap1.begin(), d_Ap1.end(),
        d_p1.begin(),
        d_p2.begin(),
        thrust::minus<COMPUTE_TYPE>()
    );
)
    /*
                CALCULATE d2
    */
time_op("d2",
    thrust::device_vector<COMPUTE_TYPE> d_d2(n);
    thrust::transform(
        d_p2.begin(),
        d_p2.end(),
        d_c3.begin(),
        d_d2.begin(),
        d2_trans()
    );
)
    /*
                CALCULATE d3
    */
time_op("d3",
    thrust::device_vector<COMPUTE_TYPE> d_d3(n);
    thrust::transform(
        d_p1.begin(),
        d_p1.end(),
        d_c3.begin(),
        d_d3.begin(),
        d3_trans()
    );
)

    hipDeviceSynchronize();

time_op("return vectors creation",
    thrust::host_vector<thrust::device_vector<COMPUTE_TYPE>> return_vector(4);
    return_vector.push_back(std::move(d_p1));
    return_vector.push_back(std::move(d_p2));
    return_vector.push_back(std::move(d_d3));
    return_vector.push_back(std::move(d_c3));
)
    return return_vector;
}

int main(int argc, char *argv[])
{
    std::chrono::system_clock::time_point start;
    std::chrono::system_clock::time_point end;

    if(argc != 2) {
        std::cout << "Usage: " << argv[0] << " <filename>" << std::endl;
        return 1;
    }
    std::string filename(argv[1]); 

    time_op("Loading the file",
        const h_csr h_A = loadSymmFileToCsr(filename);
    )

    std::cout << "A = \n";
    printCSR(h_A);

    time_op("Moving the matrix to the device",
        const d_csr d_A(h_A);
    )

    time_op("The whole fglt",
        thrust::host_vector<thrust::device_vector<COMPUTE_TYPE>> h_fglt = fglt(d_A);
    )
}