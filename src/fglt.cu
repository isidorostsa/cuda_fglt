#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/transform.h>

#include <cstddef>

#include "common/fileio.hpp"
#include "common/sparse_funcs.hpp"
#include "common/printing.hpp"
#include "common/device_csr_wrapper.hpp"

#define time_bench(func, name) \
    {\
    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();\
    func;\
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();\
    std::cout << "Time taken for " << #name << ": " << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << "ms" << std::endl;\
    }

#define CHECK_CUDA(call)                                               \
    {                                                                  \
        hipError_t status = (call);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            exit(1);                                                   \
        }                                                              \
    }

#define CHECK_CUSPARSE(call)                                                                           \
    {                                                                                                  \
        hipsparseStatus_t status = call;                                                                \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                                         \
        {                                                                                              \
            fprintf(stderr, "cuSparse error %s in file '%s' in line %i : %s.\n",                       \
                    hipsparseGetErrorName(status), __FILE__, __LINE__, hipsparseGetErrorString(status)); \
            exit(1);                                                                                   \
        }                                                                                              \
    }

#define computeType HIP_R_32F

void get_c3_v2_3(const h_csr &A, int *c3)
{  
  std::fill(c3, c3 + A.get_rows(), 0);

  for (int i = 0; i < A.get_rows(); i++)
  {
    const int i_nb_start = A.offsets[i];
    const int i_nb_end = A.offsets[i + 1];

    for (int i_nb_idx = i_nb_start; i_nb_idx < i_nb_end; i_nb_idx++)
    {
      const int j = A.positions[i_nb_idx];
      if (i < j) break;

      const int j_nb_start = A.offsets[j];
      const int j_nb_end = A.offsets[j + 1];

      int _i_nb_idx = i_nb_start;
      int _j_nb_idx = j_nb_start;

      while (_i_nb_idx < i_nb_end && _j_nb_idx < j_nb_end)
      {
        const int _i_nb_pos = A.positions[_i_nb_idx];
        const int _j_nb_pos = A.positions[_j_nb_idx];

        if(_i_nb_pos > i || _j_nb_pos > j) break;

        if (_i_nb_pos > _j_nb_pos)
        {
          _j_nb_idx++;
        }
        else if (_i_nb_pos < _j_nb_pos)
        {
          _i_nb_idx++;
        }
        else
        {
          c3[i]++, c3[j]++, c3[_i_nb_pos]++,  _i_nb_idx++, _j_nb_idx++;
        }
      }
    }
  }
}

struct d3_trans
{
    __thrust_exec_check_disable__
        __host__ __device__ constexpr float
        operator()(const float &lhs, const float &rhs) const
    {
        return (lhs) * (lhs - 1) / 2 -  rhs;
    }
};

struct d2_trans
{
    __thrust_exec_check_disable__
        __host__ __device__ constexpr float
        operator()(const float &lhs, const float &rhs) const
    {
        return (lhs) - 2 * rhs;
    }
};



int main(int argc, char *argv[])
{

    std::string filename(argv[1]); 
    // HOST DATA
    h_csr h_A = loadSymmFileToCsr(filename);

    thrust::host_vector<float> h_A_vals(h_A.nnz, 1.0f);

    const int n = h_A.rows;
    const int nnz = h_A.nnz;

    std::cout << "A = " << std::endl;
    printCSR(h_A.offsets, h_A.positions, h_A_vals, n, n, nnz);
    std::cout << "A Sparsity: " << (100 * (1 - ((float)h_A.nnz / ((float)(n) * (float)n)))) << "%" << std::endl;
    std::cout << "A positions: " << h_A.positions.size() << std::endl;

    // DEVICE DATA
    d_cusparse_csr d_A(h_A);

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle))
    /*

                A_2 = A*A COMPUTATION

    d_cusparse_csr d_A2 = d_cusparse_csr::multiply(d_A, d_A, handle);

    std::cout << "A2 = " << std::endl;
    printCSR(d_A2.get_offsets(), d_A2.get_positions(), d_A2.get_values(), n, n, d_A2.get_nnz());
    */

    /*

                CALCULATE p1

    */
    thrust::device_vector<float> d_p1(n);
    // make a copy of d_A_offs but with floats
    thrust::device_vector<float> d_A_offs_float(d_A.get_offsets());

    thrust::transform(
        d_A_offs_float.begin() + 1, d_A_offs_float.end(),
        d_A_offs_float.begin(), // d_A_offs_float.end()-1,
        d_p1.begin(),
        thrust::minus<float>());

    std::cout << "p1: " << d_p1;

    /*

                CALCULATE c3

    */

    thrust::host_vector<int> h_c3(n);
        get_c3_v2_3(h_A, h_c3.data());
    std::cout << "c3: " << h_c3;

        thrust::device_vector<float> d_c3 = h_c3;
    /*

                CALCULATE p2

    */
    thrust::device_vector<float> d_Ap1 = d_cusparse_csr::multiply(d_A, d_p1, handle);


    std::cout << "Ap1: " << d_Ap1;

    thrust::device_vector<float> d_p2(n);

        thrust::transform(
            d_Ap1.begin(), d_Ap1.end(),
            d_p1.begin(),
            d_p2.begin(),
            thrust::minus<float>()
        );

    std::cout << "p2: " << d_p2;
    /*

                CALCULATE d2

    */

    thrust::device_vector<float> d_d2(n);
        thrust::transform(
            d_p2.begin(),
            d_p2.end(),
            d_c3.begin(),
            d_d2.begin(),
            d2_trans()        );
    std::cout << "d2: " << d_d2;

    /*
                CALCULATE d3
    */

    thrust::device_vector<float> d_d3(n);
    thrust::transform(
        d_p1.begin(),
        d_p1.end(),
        d_c3.begin(),
        d_d3.begin(),
        d3_trans());
    std::cout << "d3: " << d_d3;

    // DONE

    std::cout << "d1 = " << d_p1;
    std::cout << "d2 = " << d_d2;
    std::cout << "d3 = " << d_d3;
    std::cout << "d4 = " << d_c3;
    return 0;
}