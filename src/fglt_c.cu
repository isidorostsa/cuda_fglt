#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// #include <time.h>
#include <chrono>

#include "common/host_structs.hpp"
#include "common/fileio.hpp"

#define TIME_OP(NAME, OP) \
      T_START = std::chrono::high_resolution_clock::now(); \
      OP; \
      T_END = std::chrono::high_resolution_clock::now(); \
      printf("%s took %f ms\n", NAME,  (double)std::chrono::duration_cast<std::chrono::microseconds>(T_END-T_START).count()/1000.0);
      // std::cout << name << " took " << std::chrono::duration_cast<std::chrono::milliseconds>(T_END-T_START).count() << " ms" << std::endl;W


__global__
void kernel_spmv(int n_vertices, int* A_offsets, int* A_positions, int *x, int *y) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < n_vertices){
    y[idx] = 0;

    for (int k = A_offsets[idx]; k < A_offsets[idx + 1]; k++) {
      y[idx] += x[A_positions[k]];
    } 
  }
}


__global__
void kernel_c3(int n_vertices, int* A_offsets, int* A_positions, int *c3) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n_vertices) {
    const int i_nb_start = A_offsets[i];
    const int i_nb_end = A_offsets[i + 1];

    for (int i_nb_idx = i_nb_start; i_nb_idx < i_nb_end; i_nb_idx++) {
      int j = A_positions[i_nb_idx];

      if(i<=j) break;

      const int j_nb_start = A_offsets[j];
      const int j_nb_end = A_offsets[j + 1];

      int _i_nb_idx = i_nb_start;
      int _j_nb_idx = j_nb_start;

      while (_i_nb_idx < i_nb_end && _j_nb_idx < j_nb_end)
      {
        if ((A_positions[_i_nb_idx] > i) || (A_positions[_j_nb_idx] > j)){
            break;
        }
        else if (A_positions[_i_nb_idx] == A_positions[_j_nb_idx])
        {
          c3[j]++;
          c3[i]++;
          c3[A_positions[_i_nb_idx]]++;
          _i_nb_idx++;
          _j_nb_idx++;
        }
        else if (A_positions[_i_nb_idx] < A_positions[_j_nb_idx])
        {
          _i_nb_idx++;
        }
        else
        {
          _j_nb_idx++;
        }
      }
    }
  }
}


__global__
void kernel_s0(int n_vertices, int* s0){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_vertices) {
    (s0)[idx] = 1;
  }
}
__global__
void kernel_s1(int n_vertices, int* A_offsets, int* s1){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_vertices) {
    s1[idx] = A_offsets[idx + 1] - A_offsets[idx];
  }
}

__global__
void kernel_s2(int n_vertices, int* s1, int* s2){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_vertices) {
    s2[idx] -= s1[idx];
  }
}

__global__
void kernel_s3(int n_vertices, int* s1, int* s3){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_vertices) {
    s3[idx] = (s1[idx] * (s1[idx] - 1)) / 2;
  }
}

__global__
void kernel_s4(int n_vertices, int* s2, int* s3, int* s4){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_vertices) {
     s2[idx] -= 2 * s4[idx];
     s3[idx] -= s4[idx];
  }
}

void fglt(h_csr* h_A){
  // struct timespec T_START, T_END;
  std::chrono::high_resolution_clock::time_point T_START, T_END;

  // Allocate device vectors
  int *d_d0, *d_d1, *d_d2, *d_d3, *d_d4;
  hipMalloc(&d_d0, h_A->get_rows() * sizeof(int));
  hipMalloc(&d_d1, h_A->get_rows() * sizeof(int));
  hipMalloc(&d_d2, h_A->get_rows() * sizeof(int));
  hipMalloc(&d_d3, h_A->get_rows() * sizeof(int));
  hipMalloc(&d_d4, h_A->get_rows() * sizeof(int));

  // Allocate host vectors
  int *h_d0, *h_d1, *h_d2, *h_d3, *h_d4;
  h_d0 = (int*)calloc(h_A->get_rows(), sizeof(int));
  h_d1 = (int*)calloc(h_A->get_rows(), sizeof(int));
  h_d2 = (int*)calloc(h_A->get_rows(), sizeof(int));
  h_d3 = (int*)calloc(h_A->get_rows(), sizeof(int));
  h_d4 = (int*)calloc(h_A->get_rows(), sizeof(int));
 
  int* d_A_offsets, *d_A_positions;
  TIME_OP("Moving the matrix to the device",
    // Send A to Device
    hipMalloc(&d_A_offsets, (h_A->get_rows() + 1)* sizeof(int));
    hipMalloc(&d_A_positions, h_A->get_nnz() * sizeof(int));
    hipMemcpy(d_A_offsets, h_A->get_offsets().data(), (h_A->get_rows() + 1)* sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A_positions, h_A->get_positions().data(), h_A->get_nnz() * sizeof(int), hipMemcpyHostToDevice);
  )


  // Run calculations on device
  int blockSize = 1024;
  int numBlocks = (h_A->get_rows() + blockSize - 1) / blockSize;

  TIME_OP("d0",
  (kernel_s0   <<<numBlocks, blockSize>>>(h_A->get_rows(), d_d0));
  hipDeviceSynchronize();
  );

  TIME_OP("d1",
  (kernel_s1   <<<numBlocks, blockSize>>>(h_A->get_rows(), d_A_offsets , d_d1));
  hipDeviceSynchronize();
  );

  TIME_OP("d2",
  (kernel_spmv <<<numBlocks, blockSize>>>(h_A->get_rows(), d_A_offsets, d_A_positions, d_d1, d_d2));
  (kernel_s2   <<<numBlocks, blockSize>>>(h_A->get_rows(), d_d1, d_d2));
  hipDeviceSynchronize();
  );

  TIME_OP("d3",
  (kernel_s3   <<<numBlocks, blockSize>>>(h_A->get_rows(), d_d1, d_d3));
  hipDeviceSynchronize();
  );
  
  TIME_OP("c3",
  (kernel_c3 <<<numBlocks, blockSize>>>(h_A->get_rows(), d_A_offsets, d_A_positions, d_d4)); 
  hipDeviceSynchronize();
  );
  
  TIME_OP("d4",
  (kernel_s4   <<<numBlocks, blockSize>>>(h_A->get_rows(), d_d2, d_d3, d_d4));
  hipDeviceSynchronize();
  );


  // Transfer results from device to host
  TIME_OP("Moving the results back to the host",
  hipMemcpy(h_d0, d_d0, (h_A->get_rows())*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_d1, d_d1, (h_A->get_rows())*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_d2, d_d2, (h_A->get_rows())*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_d3, d_d3, (h_A->get_rows())*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_d4, d_d4, (h_A->get_rows())*sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  );
  
  // // Validate Result
  // size_t s0=0, s1=0, s2=0, s3=0, s4=0;
  // for(int i=0; i<h_A->get_rows(); i++){
  //   s0 += h_d0[i];
  //   s1 += h_d1[i];
  //   s2 += h_d2[i];
  //   s3 += h_d3[i];
  //   s4 += h_d4[i];
  // }
  // printf("s0:%lu\ns1:%lu\ns2:%lu\ns3:%lu\ns4:%lu\n", s0, s1, s2, s3, s4);


  // Free device memory
  hipFree(d_d0);
  hipFree(d_d1);
  hipFree(d_d2);
  hipFree(d_d3);
  hipFree(d_d4);
  hipFree(d_A_offsets);
  hipFree(d_A_positions);

  // Free host memory
  free(h_d0);
  free(h_d1);
  free(h_d2);
  free(h_d3);
  free(h_d4);
}



int main(int argc, char *argv[]) {

  // Initialize cuda context
  hipFree(0);

  // struct timespec T_START, T_END;
  std::chrono::high_resolution_clock::time_point T_START, T_END;

  if (argc < 2) {
    fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
    exit(1);
  }

  // Read mtx file
  TIME_OP("Loading the file",
    h_csr h_A = loadSymmFileToCsr(argv[1]);
  )

  TIME_OP("The whole fglt",   
    fglt(&h_A);
  );

  return 0;
}